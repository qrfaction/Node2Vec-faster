#include "hip/hip_runtime.h"
#include "./generate_sample.h"
#include "./read_data.h"
#include "./tool.h"
#include "./env_init.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand.h>
#include <time.h>  

#define HANDLE_ERROR(err)  (HandleError(err, __FILE__, __LINE__ ))

using std::min;
using std::cout;
using std::endl;


void get_samples_batch(const size_t batchsize, const size_t len, 
						const size_t *nodes , size_t *host_walks){
	if(in_env==false){
		cout<<"Not initialized yet"<<endl;
		exit(0);
	}


	size_t *dev_walks = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_walks, len * batchsize * sizeof(size_t)));


	hiprandState *devStates;
	HANDLE_ERROR(hipMalloc((void **)&devStates, batchsize * sizeof(hiprandState)));


	size_t *dev_nodes = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_nodes, batchsize*sizeof(size_t)));
	HANDLE_ERROR(
		hipMemcpy(	dev_nodes,
					nodes,
					sizeof(size_t)*batchsize,
					hipMemcpyHostToDevice)); 


	int num_thread = min( max_threads_per_block , batchsize );
	int num_block = min( max_blocks_per_grid , (batchsize + num_thread-1)/num_thread );

	sample_generator<<<num_block,num_thread>>>(dev_walks,G, devStates,
							size_t(time(NULL)), dev_nodes, batchsize, len);

	HANDLE_ERROR(
		hipMemcpy(	host_walks,
					dev_walks,
					sizeof(size_t)*batchsize*len,
					hipMemcpyDeviceToHost)); 

	hipFree(dev_walks);
	hipFree(devStates);
	hipFree(dev_nodes);

}


void get_samples_epoch(const size_t len, size_t *host_walks){

	if(in_env==false){
		cout<<"Not initialized yet"<<endl;
		exit(0);
	}
	size_t *dev_walks = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_walks, len * N * sizeof(size_t)));

	hiprandState *devStates;
	HANDLE_ERROR(hipMalloc((void **)&devStates, N*sizeof(hiprandState)));



	size_t *h_nodes = new size_t[N];
	for(size_t i=0;i < N;++i)
		h_nodes[i] = i;
	size_t *dev_nodes = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_nodes, N*sizeof(size_t)));

	HANDLE_ERROR(
		hipMemcpy(	dev_nodes,
					h_nodes,
					sizeof(size_t)*N,
					hipMemcpyHostToDevice)); 


	int num_thread = min( max_threads_per_block , N );
	int num_block = min( max_blocks_per_grid , (N+num_thread-1)/num_thread );

	sample_generator<< < num_block,num_thread >> >(dev_walks,G, devStates,
						(size_t)(time(NULL)), dev_nodes, N, len);


	HANDLE_ERROR(
		hipMemcpy(	host_walks,
					dev_walks,
					sizeof(size_t)*N*len,
					hipMemcpyDeviceToHost)); 
	

	delete [] h_nodes;
	hipFree(dev_walks);
	hipFree(devStates);
	hipFree(dev_nodes);

}

