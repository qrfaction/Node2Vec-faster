#include "hip/hip_runtime.h"
#include "generate_sample.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <unordered_map>
#include <algorithm>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <assert.h>
#include <time.h>  
#include "tool.h"


using std::vector;
using std::unordered_map;
using std::pair;
using std::make_pair;
using std::cout;
using std::endl;



static void HandleError(hipError_t err, const char *file,int line){    
    if (err != hipSuccess){        
        printf( "%s in %s at line %d\n", hipGetErrorString(err),file,line);        
        exit(EXIT_FAILURE);
    }
}


#define HANDLE_ERROR(err)  (HandleError(err, __FILE__, __LINE__ ))



unordered_map<size_t, vector<pair<size_t,double > > >  
read_graph(std::string network_file, size_t & num_edge, const bool have_weight){


	std::ifstream embFile(network_file);
	
	if (embFile.is_open()){

		size_t x,y;
		auto rule_compare = compareByTwoKey();
		unordered_map<size_t,vector<pair<size_t,double > > > adj_list;
		if(have_weight){

			double weight;
			while(embFile>>x>>y>>weight){
				pair<size_t,double> ele = make_pair(y,1);
				auto pos = lower_bound(adj_list[x].begin(),adj_list[x].end(),
					ele,rule_compare);
				adj_list[x].insert(pos,ele);
				++num_edge;
			}

		}
		else{
			while(embFile>>x>>y){
				pair<size_t,double> ele = make_pair(y,1);
				auto pos = lower_bound(adj_list[x].begin(), adj_list[x].end(),
									ele, rule_compare);
				adj_list[x].insert(pos,ele);
				++num_edge;
			}
		}
		embFile.close();

		return adj_list;
	}
	else {

		cout<<"read error";
	    exit(0);
	}

}


void adjList2CSR(
	unordered_map<size_t, vector<pair<size_t,double> > > & adj_list,
	double *weights, size_t *col_id, size_t *row_shift ){

	size_t i=0;
	vector<pair<size_t, double> >  node_neighbors;

	for(size_t row_id=0; row_id<adj_list.size(); ++row_id){

		node_neighbors = adj_list[row_id];

		for(auto adj_node : node_neighbors){
			weights[i] = adj_node.second;
			col_id[i] = adj_node.first;
			++i;
		}
		row_shift[row_id+1] = i;
	}

}

size_t * init_graph(){

	size_t num_edge=0;
	auto adj_list = read_graph("edges.csv",num_edge,false);


	double * weights = new double[num_edge];

	size_t * col_id = new size_t[num_edge];
	size_t * row_shift = new size_t[adj_list.size()+1];
	row_shift[0] = 0;


	adjList2CSR(adj_list,weights,col_id,row_shift);


	csr_graph * host_g = new csr_graph(col_id,row_shift,weights, adj_list.size(),num_edge,1.0,1.0);
	
	

}


size_t * get_samples_batch(csr_graph * g, const size_t batchsize, const size_t len, const size_t *nodes){

	size_t *host_walks = new size_t[batchsize*len];

	size_t *dev_walks = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_walks, len * batchsize * sizeof(size_t)));

	hiprandState *devStates;
	HANDLE_ERROR(hipMalloc((void **)&devStates, batchsize * sizeof(hiprandState)));

	size_t blocks = min(batchsize,(size_t)512);

	sample_generator<<<blocks,1>>>(dev_walks, g, devStates,
							unsigned(time(NULL)), nodes, batchsize, len);

	HANDLE_ERROR(
		hipMemcpy(	host_walks,
					dev_walks,
					sizeof(size_t)*batchsize*len,
					hipMemcpyDeviceToHost)); 

	hipFree(dev_walks);
	hipFree(devStates);

	return host_walks;
}


size_t * get_samples_epoch(csr_graph * g, const size_t len){

	size_t batchsize = min((size_t)512,g->num_node);

	size_t *host_walks = new size_t[(g->num_node)*len];

	size_t *dev_walks = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_walks, len * batchsize * sizeof(size_t)));

	hiprandState *devStates;
	HANDLE_ERROR(hipMalloc((void **)&devStates, batchsize * sizeof(hiprandState)));

	size_t *nodes = new size_t[batchsize];

	for(size_t i=0;i < g->num_node; i+=batchsize){

		for(size_t j=0;j<batchsize;++j)
			nodes[j] = i+j;

		sample_generator<<<batchsize,1>>>(dev_walks, g, devStates,
							unsigned(time(NULL)), nodes, batchsize, len);

		HANDLE_ERROR(
			hipMemcpy(	&host_walks[i],
						dev_walks,
						sizeof(size_t)*batchsize*len,
						hipMemcpyDeviceToHost)); 

	}

	

	hipFree(dev_walks);
	hipFree(devStates);

	return host_walks;
}